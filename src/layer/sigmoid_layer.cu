#include "hip/hip_runtime.h"
#include "sigmoid_layer.hpp"

__device__ auto d_sigmoid_activation(float x) -> float
{
    return 1.0f / (1 + expf(-x));
}

__global__ auto d_forward_propagation(float* Z
                                     ,float* A
                                     ,int Z_x
                                     ,int Z_y) -> void
{
    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(thread_idx < (Z_x * Z_y))
        A[thread_idx] = d_sigmoid_activation(Z[thread_idx]);
}

__global__ auto d_backward_propagation(float* Z
                                      ,float* A_error
                                      ,float* Z_error
                                      ,int Z_x
                                      ,int Z_y) -> void
{
    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(thread_idx < (Z_x * Z_y))
        Z_error[thread_idx] = A_error[thread_idx] * d_sigmoid_activation(Z[thread_idx]) * (1 - d_sigmoid_activation(Z[thread_idx]));
}

SigmoidLayer::SigmoidLayer(std::string title)
{
    this->title = title;
}

auto SigmoidLayer::forward_propagation(Tensor& Z) -> Tensor&
{
    this->Z = Z;
    A.allocate_memory_if_not_allocated(Z.dimension);
    dim3 block_size(256);
    dim3 block_count((Z.dimension.x * Z.dimension.y + block_size.x - 1) / block_size.x);
    d_forward_propagation<<<block_count, block_size>>>(Z.device_data.get()
                                                      ,A.device_data.get()
                                                      ,Z.dimension.x
                                                      ,Z.dimension.y);
    return A;
}

auto SigmoidLayer::backward_propagation(Tensor& A_error, float learning_rate) -> Tensor&
{
    Z_error.allocate_memory_if_not_allocated(Z.dimension);\
    dim3 block_size(256);
    dim3 block_count((Z.dimension.x * Z.dimension.y + block_size.x - 1) / block_size.x);
    d_backward_propagation<<<block_count, block_size>>>(Z.device_data.get()
                                                       ,A_error.device_data.get()
                                                       ,Z_error.device_data.get()
                                                       ,Z.dimension.x
                                                       ,Z.dimension.y);          
    return Z_error;
}

int main(){return 1;}